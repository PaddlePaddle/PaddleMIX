#include <hip/hip_runtime.h>
#include <stdint.h>
#include <assert.h>

// launcher for: split_concat_fp16_2048_kernel_2048_warps4xstages4
hipError_t split_concat_fp16_2048_kernel_9dc56806_0d1d2d3d4d56d78d(hipStream_t stream, hipDeviceptr_t out0, hipDeviceptr_t out1, hipDeviceptr_t out2, hipDeviceptr_t qkv, hipDeviceptr_t eqkv, int32_t batch, int32_t seq_qkv, int32_t seq_eqkv, int32_t output_hidden);

hipError_t split_concat_fp16_2048_kernel_2048_warps4xstages4(hipStream_t stream, hipDeviceptr_t out0, hipDeviceptr_t out1, hipDeviceptr_t out2, hipDeviceptr_t qkv, hipDeviceptr_t eqkv, int32_t batch, int32_t seq_qkv, int32_t seq_eqkv, int32_t output_hidden){
  if ((out0 % 16 == 0) && (out1 % 16 == 0) && (out2 % 16 == 0) && (qkv % 16 == 0) && (eqkv % 16 == 0) && (seq_qkv % 16 == 0) && (output_hidden % 16 == 0))
    return split_concat_fp16_2048_kernel_9dc56806_0d1d2d3d4d56d78d(stream, out0, out1, out2, qkv, eqkv, batch, seq_qkv, seq_eqkv, output_hidden);

  return hipErrorInvalidValue;
}

// load for: split_concat_fp16_2048_kernel_2048_warps4xstages4
void load_split_concat_fp16_2048_kernel_9dc56806_0d1d2d3d4d56d78d();
void load_split_concat_fp16_2048_kernel_2048_warps4xstages4() {
  load_split_concat_fp16_2048_kernel_9dc56806_0d1d2d3d4d56d78d();
}

// unload for: split_concat_fp16_2048_kernel_2048_warps4xstages4
void unload_split_concat_fp16_2048_kernel_9dc56806_0d1d2d3d4d56d78d();
void unload_split_concat_fp16_2048_kernel_2048_warps4xstages4() {
  unload_split_concat_fp16_2048_kernel_9dc56806_0d1d2d3d4d56d78d();
}

typedef hipError_t (*kernel_func_t)(hipStream_t stream, hipDeviceptr_t out0, hipDeviceptr_t out1, hipDeviceptr_t out2, hipDeviceptr_t qkv, hipDeviceptr_t eqkv, int32_t batch, int32_t seq_qkv, int32_t seq_eqkv, int32_t output_hidden);
kernel_func_t split_concat_fp16_2048_kernel_kernels[] = {
  split_concat_fp16_2048_kernel_2048_warps4xstages4,
};

int split_concat_fp16_2048_kernel_get_num_algos(void){
  return (int)(sizeof(split_concat_fp16_2048_kernel_kernels) / sizeof(split_concat_fp16_2048_kernel_kernels[0]));
}

hipError_t split_concat_fp16_2048_kernel(hipStream_t stream, hipDeviceptr_t out0, hipDeviceptr_t out1, hipDeviceptr_t out2, hipDeviceptr_t qkv, hipDeviceptr_t eqkv, int32_t batch, int32_t seq_qkv, int32_t seq_eqkv, int32_t output_hidden, int algo_id){
  assert (algo_id < (int)sizeof(split_concat_fp16_2048_kernel_kernels));
  return split_concat_fp16_2048_kernel_kernels[algo_id](stream, out0, out1, out2, qkv, eqkv, batch, seq_qkv, seq_eqkv, output_hidden);
}

void load_split_concat_fp16_2048_kernel(void){
  load_split_concat_fp16_2048_kernel_2048_warps4xstages4();
}

void unload_split_concat_fp16_2048_kernel(void){
  unload_split_concat_fp16_2048_kernel_2048_warps4xstages4();
}


hipError_t split_concat_fp16_2048_kernel_default(hipStream_t stream, hipDeviceptr_t out0, hipDeviceptr_t out1, hipDeviceptr_t out2, hipDeviceptr_t qkv, hipDeviceptr_t eqkv, int32_t batch, int32_t seq_qkv, int32_t seq_eqkv, int32_t output_hidden){
  return split_concat_fp16_2048_kernel(stream, out0, out1, out2, qkv, eqkv, batch, seq_qkv, seq_eqkv, output_hidden, 0);
}
