#include <vector>
#include <map>
#include "triton_fused_adaLN_scale_residual_fp16_2048_0_0_kernel.h"
#include "paddle/extension.h"

std::map<std::vector<int>, int> map_problem_triton_fused_adaLN_scale_residual_fp16_2048_0_0;

hipDeviceptr_t get_tensor_ptr(const paddle::Tensor& input){
  if (input.type() == paddle::DataType::FLOAT16) {
    return (hipDeviceptr_t)(input.data<phi::dtype::float16>());
  } else if (input.type() == paddle::DataType::BFLOAT16) {
    return (hipDeviceptr_t)(input.data<phi::dtype::bfloat16>());
  } else if (input.type() == paddle::DataType::INT32) {
    return (hipDeviceptr_t)(input.data<int>());
  } else if (input.type() == paddle::DataType::FLOAT32) {
    return (hipDeviceptr_t)(input.data<float>());
  } else if (input.type() == paddle::DataType::UINT8) {
    return (hipDeviceptr_t)(input.data<uint8_t>());
  } else if (input.type() == paddle::DataType::INT8) {
    return (hipDeviceptr_t)(input.data<int8_t>());
  } else {
    assert(false);
    return (hipDeviceptr_t)(nullptr);
  }
} 


std::vector<paddle::Tensor> triton_fused_adaLN_scale_residual_fp16_2048_0_0_func(
    const paddle::Tensor &x,
    const paddle::Tensor &mha_out,
    const paddle::Tensor &gate_msa,
    const paddle::Tensor &scale_mlp,
    const paddle::Tensor &shift_mlp,
    paddle::optional<paddle::Tensor> &weight,
    paddle::optional<paddle::Tensor> &bias,
    float epsilon) {
  int M = x.dims()[0] * x.dims()[1];
  int N = x.dims()[2];
  int seq_size = x.dims()[1];
  auto resi_out = paddle::empty(x.shape(), x.dtype(), x.place());
  auto adaLN_out = paddle::empty(x.shape(), x.dtype(), x.place());

  auto x_ptr = get_tensor_ptr(x);
  auto mha_out_ptr = get_tensor_ptr(mha_out);
  auto resi_out_ptr = get_tensor_ptr(resi_out);
  auto adaLN_out_ptr = get_tensor_ptr(adaLN_out);
  auto gate_msa_ptr = get_tensor_ptr(gate_msa);
  auto scale_mlp_ptr = get_tensor_ptr(scale_mlp);
  auto shift_mlp_ptr = get_tensor_ptr(shift_mlp);
  hipDeviceptr_t weight_ptr = (hipDeviceptr_t)(nullptr);
  if (weight) {
    weight_ptr = get_tensor_ptr(*weight);
  }
  hipDeviceptr_t bias_ptr = (hipDeviceptr_t)(nullptr);
  if (bias) {
    bias_ptr = get_tensor_ptr(*bias);
  }
  auto  run_stream = adaLN_out.stream();

  std::vector<int> problem_size = {M};
  auto run_triton_kernel = [&](int algo_id) -> hipError_t{
      return triton_fused_adaLN_scale_residual_fp16_2048_0_0_kernel(run_stream,
                                               x_ptr,mha_out_ptr,gate_msa_ptr,scale_mlp_ptr,shift_mlp_ptr,weight_ptr,bias_ptr,resi_out_ptr,adaLN_out_ptr,M,N,seq_size,epsilon,

                                               algo_id);
  };

  if (!map_problem_triton_fused_adaLN_scale_residual_fp16_2048_0_0.count(problem_size)) {
    std::cout << "we are tuning for triton_fused_adaLN_scale_residual_fp16_2048_0_0 which key is: {";
    for (int i = 0; i < problem_size.size(); i++) {
        std::cout << problem_size[i] << ", ";
    }
    std::cout << "}" << std::endl;

    float min_time = 10000.f;
    int select_id = -1;
    constexpr int WARMUP = 5;
    constexpr int REPEAT = 10;

    for (int algo_id = 0; algo_id < triton_fused_adaLN_scale_residual_fp16_2048_0_0_kernel_get_num_algos(); ++algo_id) {
        hipEvent_t beg[REPEAT];
        hipEvent_t end[REPEAT];
        float elapsed_times[REPEAT];

        auto status = hipSuccess;

        for (int ii = 0; ii < WARMUP + REPEAT; ii++) {
            int repeat_id = ii - WARMUP;

            if (repeat_id >= 0) {
                (hipEventCreate(beg + repeat_id));
                (hipEventCreate(end + repeat_id));
                (hipEventRecord(beg[repeat_id]));
            }

            auto flush_l2_cache = paddle::full(
                {10 * 1024 * 1024}, 0, paddle::DataType::INT32, x.place());
            // std::cout << &flush_l2_cache  << std::endl;
            // this is used when out is need to be reset to zero, such as split-k gemm.
            ;

            status = run_triton_kernel(algo_id);
            // assert(status == hipSuccess);

            if (repeat_id >= 0) {
                (hipEventRecord(end[repeat_id]));
                (hipEventSynchronize(end[repeat_id]));
                (hipEventElapsedTime(
                    elapsed_times + repeat_id, beg[repeat_id], end[repeat_id]));
            }
        }

        float avg_elapsed_time = 0.f;
        for (int ii = 0; ii < REPEAT; ++ii) {
            avg_elapsed_time += elapsed_times[ii];
        }

        std::cout << "algo id " << algo_id << " costs " << avg_elapsed_time << " ms" << std::endl;

        if (avg_elapsed_time < min_time && status == hipSuccess) {
            min_time = avg_elapsed_time;
            select_id = algo_id;
        }
    }

    map_problem_triton_fused_adaLN_scale_residual_fp16_2048_0_0[problem_size] = select_id;
    std::cout << "select algo id: " << select_id << std::endl;
    ;
  }

  if (map_problem_triton_fused_adaLN_scale_residual_fp16_2048_0_0.count(problem_size)) {
    int algo_id = map_problem_triton_fused_adaLN_scale_residual_fp16_2048_0_0[problem_size];
    auto status = run_triton_kernel(algo_id);
    assert(status == hipSuccess);
  }

    return {resi_out, adaLN_out};
}

std::vector<std::vector<int64_t>> triton_fused_adaLN_scale_residual_fp16_2048_0_0_InferShape(
        const std::vector<int64_t>& A_shape) {
  return {A_shape, A_shape};
}

std::vector<paddle::DataType> triton_fused_adaLN_scale_residual_fp16_2048_0_0_InferDtype(const paddle::DataType& A_dtype) {
    return {A_dtype, A_dtype};
}

PD_BUILD_OP(triton_fused_adaLN_scale_residual_fp16_2048_0_0)
    .Inputs({"x", "mha_out", "gate_msa", "scale_mlp", "shift_mlp", paddle::Optional("weight"), paddle::Optional("bias")})
    .Outputs({"resi_out", "adaLN_out"})
    .SetKernelFn(PD_KERNEL(triton_fused_adaLN_scale_residual_fp16_2048_0_0_func))
    .Attrs({"epsilon: float"})
    .SetInferDtypeFn(PD_INFER_DTYPE(triton_fused_adaLN_scale_residual_fp16_2048_0_0_InferDtype))
    .SetInferShapeFn(PD_INFER_SHAPE(triton_fused_adaLN_scale_residual_fp16_2048_0_0_InferShape));
